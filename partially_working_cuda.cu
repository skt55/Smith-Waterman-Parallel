#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <bits/stdc++.h>
#include "smith-waterman.hpp"

using namespace std;

__global__ void smith_waterman_kernel(
        const char* __restrict__ seq1,
        const char* __restrict__ seq2,
        int* __restrict__ score,
        int* __restrict__ maxScore,
        int* __restrict__ maxI,
        int* __restrict__ maxJ,
        int size1,
        int size2,
        int match,
        int mismatch,
        int gap,
        int diag)
{
    // Compute the thread's position in the anti-diagonal
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int i = diag - idx;
    int j = idx + 1;

    if (i < 1 || i > size1 || j < 1 || j > size2)
        return;

    // Linear index in the score matrix
    int index = i * (size2 + 1) + j;// considering row major ordering
    int index_diag = (i - 1) * (size2 + 1) + (j - 1);
    int index_up = (i - 1) * (size2 + 1) + j;
    int index_left = i * (size2 + 1) + (j - 1);

    char a = seq1[i - 1]; // Adjusting for 0-based indexing
    char b = seq2[j - 1];

    int matchScore = (a == b) ? match : mismatch;

    int score_diag = score[index_diag] + matchScore;
    int score_up = score[index_up] + gap;
    int score_left = score[index_left] + gap;

    int cellScore = max(0, max(score_diag, max(score_up, score_left)));
    score[index] = cellScore;

    // Update maximum score using atomic operations
    if (cellScore > 0)
    {
        int oldMax = atomicMax(maxScore, cellScore);
        if (cellScore > oldMax)
        {
            atomicExch(maxI, i);
            atomicExch(maxJ, j);
        }
    }
}




std::pair<std::string, std::string> smithWaterman(const char *seq1, size_t size1, const char *seq2, size_t size2)
{

    
    int match = 2;     // Score for a match
    int mismatch = -1; // Score for a mismatch
    int gap = -1;      // Score for a gap

    std::vector<int> hscore((size1+1)*(size2+1),0);
    
    //memory allocation
    
    char *cuda_seq1, *cuda_seq2;
    int *maxScore, *maxI, *maxJ, *score;
    int hmaxScore, hmaxI, hmaxJ;
    hipMalloc((void**)&cuda_seq1, size1 * sizeof(char));
    hipMalloc((void**)&cuda_seq2, size2 * sizeof(char));
    hipMalloc((void**)&score, (size1 + 1) * (size2 + 1) * sizeof(int));
    hipMalloc((void**)&maxScore, sizeof(int));
    hipMalloc((void**)&maxI, sizeof(int));
    hipMalloc((void**)&maxJ, sizeof(int));


    //copy data to the gpu
    hipMemcpy(cuda_seq1, seq1, size1 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(cuda_seq2, seq2, size2 * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(score, 0, (size1 + 1) * (size2 + 1) * sizeof(int));
    hipMemset(maxScore, 0, sizeof(int));
    hipMemset(maxI, 0, sizeof(int));
    hipMemset(maxJ, 0, sizeof(int));
    
    int total_diagonals = size1 + size2 - 1;

    for (int diag = 1; diag <= total_diagonals; ++diag)
    {
        // Explicitly cast size1 and size2 to int
        int elements_in_diag = min(diag, min(static_cast<int>(size1), static_cast<int>(size2)));

        int threads_per_block = 256;
        //total threads shoudl be atleast equal to number of elements in the digonal
        int blocks = (elements_in_diag + threads_per_block - 1) / threads_per_block;

        smith_waterman_kernel<<<blocks, threads_per_block>>>(
                cuda_seq1, cuda_seq2, score, maxScore, maxI, maxJ,
                size1, size2, match, mismatch, gap, diag);
        // all kernels should finish before moving to the next diagonal
        hipDeviceSynchronize();
    }

    hipMemcpy(hscore.data(),score, (size1 + 1) * (size2 + 1) * sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(&hmaxScore, maxScore,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(&hmaxI, maxI, sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(&hmaxJ,maxJ, sizeof(int),hipMemcpyDeviceToHost);

    // Backtrack to find the aligned sequences
    std::string alignedSeq1, alignedSeq2;
    size_t i = hmaxI, j = hmaxJ;
    cout<<"in backtracking"<< i << "and" << j;
    while (i > 0 && j > 0 && (hscore[i*(size2+1)+ j] > 0))
    {
        if (seq1[i - 1] == seq2[j - 1])
        {
            alignedSeq1 += seq1[i - 1];
            alignedSeq2 += seq2[j - 1];
            i--;
            j--;
        }
        else if (hscore[i*(size2+1)+ j] == hscore[(i-1)*(size2+1)+ j] + gap)
        {
            alignedSeq1 += seq1[i - 1];
            alignedSeq2 += '-';
            i--;
        }
        else
        {
            alignedSeq1 += '-';
            alignedSeq2 += seq2[j - 1];
            j--;
        }
    }

    // Reverse the aligned sequences
    std::reverse(alignedSeq1.begin(), alignedSeq1.end());
    std::reverse(alignedSeq2.begin(), alignedSeq2.end());
    hipFree(score);
    hipFree(maxScore);
    hipFree(maxI);
    hipFree(maxJ);
    return {alignedSeq1, alignedSeq2}; // Return the aligned sequences
}
