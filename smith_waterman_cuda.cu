#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <bits/stdc++.h>

#define MATCH_SCORE 2
#define MISMATCH_SCORE -1
#define GAP_PENALTY -1

//variable names taken from GPU computing gems
//this does a row of the matrix

//matrices is row major
__global__ void rowscan(int *H, int* E_bar, int *F, size_t row_length, int curr_row, char* seq1, char* seq2) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    //remember that col 0 is all 0
    if (i > 0 && i <= row_length) {
        // as in the book
        int matchScore = (seq1[i - 1] == seq2[curr_row - 1]) ? MATCH_SCORE : MISMATCH_SCORE;

        //F part
        int F_up = F[i + (curr_row - 1) * row_length];
        int H_up = H[i + (curr_row - 1) * row_length];
        int F_me = max(F_up, H_up - 1) - 1;
        F[i + (curr_row) * row_length] = F_me;

        //Hbar
        int H_diag = H[i - 1 + (curr_row - 1) * row_length];
        int H_bar_me = max(max(H_diag + matchScore, F_me), 0);


        //START OF E_BAR PART
        //MAX PREFIIX SCAN

        //this is as wide as the row
        extern __shared__ int shared_data[];

        int tid = threadIdx.x;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        
        //this will not hold the zero in col 0
        shared_data[i - 1] = H[i];
        
        __syncthreads();

        // Perform the prefix maximum operation in shared memory
        for (int offset = 1; offset < blockDim.x; offset *= 2) {
            int temp = 0;
            if (i - 1 >= offset) {
                temp = shared_data[i - offset - 1];
            }
            __syncthreads();
            shared_data[i - 1] = max(shared_data[i - 1], temp);
            __syncthreads();
        }

        E_bar[i + (curr_row) * row_length] = shared_data[i - 1];
        // Write the result to global memory


        //finally H
        int E_bar_me = E_bar[i + (curr_row) * row_length];
        int H_me = max(H_bar_me, E_bar_me - 1);

        H[i + (curr_row) * row_length] = H_me; 
    }
}

/*
__global__ void maxPrefixScan_Ebar(int* input, int* output, int N) {
    extern __shared__ int shared_data[];

        int tid = threadIdx.x;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < N) {
            shared_data[tid] = input[idx];
        }
        __syncthreads();

        // Perform the prefix maximum operation in shared memory
        for (int offset = 1; offset < blockDim.x; offset *= 2) {
            int temp = 0;
            if (tid >= offset) {
                temp = shared_data[tid - offset];
            }
            __syncthreads();
            shared_data[tid] = max(shared_data[tid], temp);
            __syncthreads();
        }

        // Write the result to global memory
        if (idx < N) {
            output[idx] = shared_data[tid];
        }
}
*/

std::pair<std::string, std::string> smithWaterman(const char *seq1, size_t size1, const char *seq2, size_t size2) {
    // Allocate memory on the device
    int *d_H, *d_E, *d_F, *d_S, *d_Gs, *d_Ge;
    hipMalloc(&d_H, sizeof(int) * (size1 + 1) * (size2 + 1));
    hipMalloc(&d_E, sizeof(int) * (size1 + 1) * (size2 + 1));
    hipMalloc(&d_F, sizeof(int) * (size1 + 1) * (size2 + 1);

    // Copy data to device
    hipMemset(d_H, 0, sizeof(int) * (size1 + 1) * (size2 + 1));
    hipMemset(d_E, 0, sizeof(int) * (size1 + 1) * (size2 + 1));
    hipMemset(d_F, 0, sizeof(int) * (size1 + 1) * (size2 + 1));


    // Define block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((n + blockDim.x - 1) / blockDim.x, (m + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    for (int row = 1; row <= size2; row++ ) {
        rowscan<<<gridDim, blockDim>>>(d_H, d_E, d_F, d_S, d_Gs, d_Ge, n, m);
    }

    // Copy result back to host
    int* H = (int*)malloc(sizeof(int) * (size1 + 1) * (size2 + 1));
    hipMemcpy(H, d_H, sizeof(int) * (size1 + 1) * (size2 + 1), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_H);
    hipFree(d_E);
    hipFree(d_F);

    // Backtrack to find the aligned sequences
    std::string alignedSeq1, alignedSeq2;
    size_t i = maxI, j = maxJ;

    while (i > 0 && j > 0 && score[i][j] > 0)
    {
        if (seq1[i - 1] == seq2[j - 1])
        {
            alignedSeq1 += seq1[i - 1];
            alignedSeq2 += seq2[j - 1];
            i--;
            j--;
        }
        else if (score[i][j] == score[i - 1][j] + gap)
        {
            alignedSeq1 += seq1[i - 1];
            alignedSeq2 += '-';
            i--;
        }
        else
        {
            alignedSeq1 += '-';
            alignedSeq2 += seq2[j - 1];
            j--;
        }
    }

    // Reverse the aligned sequences
    std::reverse(alignedSeq1.begin(), alignedSeq1.end());
    std::reverse(alignedSeq2.begin(), alignedSeq2.end());

    return {alignedSeq1, alignedSeq2}; // Return the aligned sequences
}