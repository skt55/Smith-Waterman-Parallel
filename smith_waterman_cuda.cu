#include "hip/hip_runtime.h"
#include <chrono>
#include <algorithm>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// This kenel fills the dp matrix based on 3 of its neighbors and penalties
__global__ void smith_waterman_kernel_optimized(
    const char *__restrict__ seq1,
    const char *__restrict__ seq2,
    int *__restrict__ score,
    int size1,
    int size2,
    int match,
    int mismatch,
    int gap,
    int diag)
{
    // shared memory used by threads in a block
    extern __shared__ int shared_score[];
    // current thread parameters
    int thread_id = threadIdx.x;
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned long long i = diag - global_idx;
    unsigned long long j = global_idx + 1;

    // checking bounds
    if (i < 1 || i > size1 || j < 1 || j > size2) return;

    // Compute indices
    unsigned long long index = i * (size2 + 1) + j;
    unsigned long long index_diag = (i - 1) * (size2 + 1) + (j - 1);
    unsigned long long index_up = (i - 1) * (size2 + 1) + j;
    unsigned long long index_left = i * (size2 + 1) + (j - 1);

    // checking miss match or match
    char a = seq1[i - 1];
    char b = seq2[j - 1];
    int matchScore = (a == b) ? match : mismatch;

    // Use shared memory to store neighboring values
    shared_score[thread_id] = score[index_diag];

    __syncthreads();

    // computing all the possibilites, removed conditional statements for speed up
    int score_diag = shared_score[thread_id] + matchScore;
    int score_up = score[index_up] + gap;
    int score_left = score[index_left] + gap;

    //updating the matrix
    int cellScore = max(0, max(score_diag, max(score_up, score_left)));
    score[index] = cellScore;
}

std::pair<std::string, std::string> smithWaterman(
    const char *seq1,
    size_t size1,
    const char *seq2,
    size_t size2)
{
    // penalties
    int match = 2, mismatch = -1, gap = -1;
    std::vector<int> hscore((size1 + 1) * (size2 + 1), 0);

    char *cuda_seq1, *cuda_seq2;
    int *cuda_score;
    auto start1 = std::chrono::high_resolution_clock::now();

    // assiging memory on the device
    hipMalloc((void **)&cuda_seq1, size1 * sizeof(char));
    hipMalloc((void **)&cuda_seq2, size2 * sizeof(char));
    hipMalloc((void **)&cuda_score, (size1 + 1) * (size2 + 1) * sizeof(int));
    auto start2 = std::chrono::high_resolution_clock::now();

    // sending sequences to the device
    hipMemcpy(cuda_seq1, seq1, size1 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(cuda_seq2, seq2, size2 * sizeof(char), hipMemcpyHostToDevice);
    hipMemset(cuda_score, 0, (size1 + 1) * (size2 + 1) * sizeof(int));
    auto start3 = std::chrono::high_resolution_clock::now();

    // moving in wave-front (anti-diagonal order)
    int total_diagonals = size1 + size2 - 1;
    int threads_per_block = 1024; 
    // shared memory size to be used by each thread in a block
    int shared_mem_size = threads_per_block * sizeof(int);

    for (int diag = 1; diag <= total_diagonals; ++diag)
    {
        int elements_in_diag = min(diag, min(static_cast<int>(size1), static_cast<int>(size2)));
        // number of blocks required for the given diagonal
        int blocks = (elements_in_diag + threads_per_block - 1) / threads_per_block;

        smith_waterman_kernel_optimized<<<blocks, threads_per_block, shared_mem_size>>>(
            cuda_seq1, cuda_seq2, cuda_score,
            size1, size2, match, mismatch, gap, diag);
        // this sync ensures , nxt diagonal is solved only when prev is completed
        hipDeviceSynchronize();
    }
    auto start4 = std::chrono::high_resolution_clock::now();

    // copying data back from the device to the host
    hipMemcpy(hscore.data(), cuda_score, (size1 + 1) * (size2 + 1) * sizeof(int), hipMemcpyDeviceToHost);
    auto start5 = std::chrono::high_resolution_clock::now();

    
    int maxI = 0,maxJ;
    int count = 0;
    for(int row = 0;row<=size1;++row){
        for(int col = 0;col<=size2;++col){
            int curr_element = hscore[row*(size1+1) + col];
            if(count<curr_element){
                maxI = row;
                maxJ = col;
                count = curr_element;
            }
         
        }
       
    }
    auto start6 = std::chrono::high_resolution_clock::now();

    // Backtrack to find the aligned sequences
    std::string alignedSeq1, alignedSeq2;
    int i = maxI, j = maxJ;
    while (i > 0 && j > 0 && (hscore[i*(size2+1)+ j] > 0))
    {
        if (seq1[i - 1] == seq2[j - 1])
        {
            alignedSeq1 += seq1[i - 1];
            alignedSeq2 += seq2[j - 1];
            i--;
            j--;
        }
        else if (hscore[i*(size2+1)+ j] == hscore[(i-1)*(size2+1)+ j] + gap)
        {
            alignedSeq1 += seq1[i - 1];
            alignedSeq2 += '-';
            i--;
        }
        else
        {
            alignedSeq1 += '-';
            alignedSeq2 += seq2[j - 1];
            j--;
        }
    }

    auto start7 = std::chrono::high_resolution_clock::now();

    hipFree(cuda_seq1);
    hipFree(cuda_seq2);
    hipFree(cuda_score);   
    auto start8 = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> duration = start2 - start1;
    std::cout << "Memory1:           " << duration.count() << " seconds" << std::endl;

    std::chrono::duration<double> duration1 = start3 - start2;
    std::cout << "Memset/cpy:        " << duration1.count() << " seconds" << std::endl;

    std::chrono::duration<double> duration2 = start4 - start3;
    std::cout << "kernel time taken: " << duration2.count() << " seconds" << std::endl;

    std::chrono::duration<double> duration3 = start5 - start4;
    std::cout << "cpy2 time taken:   " << duration3.count() << " seconds" << std::endl;

    std::chrono::duration<double> duration4 = start6 - start5;
    std::cout << "max indices fetch: " << duration4.count() << " seconds" << std::endl;

    std::chrono::duration<double> duration5 = start7 - start6;
    std::cout << "backtrack time:    " << duration5.count() << " seconds" << std::endl;

     std::chrono::duration<double> duration6 = start8 - start7;
    std::cout << "cuda free:        " << duration6.count() << " seconds" << std::endl;
    // Reverse the aligned sequences
    std::reverse(alignedSeq1.begin(), alignedSeq1.end());
    std::reverse(alignedSeq2.begin(), alignedSeq2.end());
    std::cout << "Total cuda time is " << duration.count() + duration1.count() + duration2.count() + duration3.count() + duration6.count()<< "seconds" << std::endl;
    return {alignedSeq1, alignedSeq2}; // Return the aligned sequences
}



    
